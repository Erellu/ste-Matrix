#include "CUDA_global.h"


//void cublasErrchk(hipblasStatus_t code){
//    cublasAssert(code , __FILE__ , __LINE__);
//}

void cublasAssert(hipblasStatus_t code, const char *file, int line){

    if(code != HIPBLAS_STATUS_SUCCESS){
        std::cerr <<  "CUBLAS error.\nError code: ";

        switch(code){
            case HIPBLAS_STATUS_SUCCESS:{std::cerr << "HIPBLAS_STATUS_SUCCESS."; break;}

            case HIPBLAS_STATUS_NOT_INITIALIZED:{std::cerr << "HIPBLAS_STATUS_NOT_INITIALIZED."; break;}

            case HIPBLAS_STATUS_ALLOC_FAILED:{std::cerr << "HIPBLAS_STATUS_ALLOC_FAILED."; break;}

            case HIPBLAS_STATUS_INVALID_VALUE:{std::cerr << "HIPBLAS_STATUS_INVALID_VALUE."; break;}

            case HIPBLAS_STATUS_ARCH_MISMATCH:{std::cerr << "HIPBLAS_STATUS_ARCH_MISMATCH."; break;}

            case HIPBLAS_STATUS_MAPPING_ERROR:{std::cerr << "HIPBLAS_STATUS_MAPPING_ERROR."; break;}

            case HIPBLAS_STATUS_EXECUTION_FAILED:{std::cerr << "HIPBLAS_STATUS_EXECUTION_FAILED."; break;}

            case HIPBLAS_STATUS_INTERNAL_ERROR:{std::cerr << "HIPBLAS_STATUS_INTERNAL_ERROR."; break;}

            case HIPBLAS_STATUS_NOT_SUPPORTED:{std::cerr << "HIPBLAS_STATUS_NOT_SUPPORTED."; break;}

            case HIPBLAS_STATUS_UNKNOWN:{std::cerr << "HIPBLAS_STATUS_UNKNOWN."; break;}

            default:{std::cerr << "<unknown>."; break;}

        }

        std::cerr << "\nFile: "<< file << "\n";
        std::cerr << "Line: "<< line <<std::endl;

        exit(EXIT_FAILURE);

    }


}




void gpuAssert(hipError_t code, const char *file, int line){

   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      //if (abort) exit(code);
   }
   
}

//void gpuErrchk(hipError_t code){gpuAssert(code , __FILE__ , __LINE__);}


