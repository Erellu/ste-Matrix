/**

                     ste::Matrix class

    @brief This CUDA source file contains convenience functions for assertions in CUDA.

    @copyright     Copyright (C) <2020-2021>  DUHAMEL Erwan

                        BSD-2 License

    All rights reserved.

    Redistribution and use in source and binary forms, with or without modification,
    are permitted provided that the following conditions are met:

        * Redistributions of source code must retain the above copyright notice,
          this list of conditions and the following disclaimer.
        * Redistributions in binary form must reproduce the above copyright notice,
          this list of conditions and the following disclaimer in the documentation
          and/or other materials provided with the distribution.

    THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
    "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
    LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
    A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER
    OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
    EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
    PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
    PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
    LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
    NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
    SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.


    @authors DUHAMEL Erwan (erwanduhamel@outlook.com)            -- Developper / Tester
             SOUDIER Jean  (jean.soudier@insa-strasbourg.fr)     -- Tester
*/


#include "CUDA_global.h"

namespace ste {

void CUBLAs_assert(const hipblasStatus_t &code, const char *file, const size_t &line){

    if(code != HIPBLAS_STATUS_SUCCESS){
        std::cerr <<  "CUBLAS error.\nError code: ";

        switch(code){
            case HIPBLAS_STATUS_SUCCESS:{std::cerr << "HIPBLAS_STATUS_SUCCESS."; break;}

            case HIPBLAS_STATUS_NOT_INITIALIZED:{std::cerr << "HIPBLAS_STATUS_NOT_INITIALIZED."; break;}

            case HIPBLAS_STATUS_ALLOC_FAILED:{std::cerr << "HIPBLAS_STATUS_ALLOC_FAILED."; break;}

            case HIPBLAS_STATUS_INVALID_VALUE:{std::cerr << "HIPBLAS_STATUS_INVALID_VALUE."; break;}

            case HIPBLAS_STATUS_ARCH_MISMATCH:{std::cerr << "HIPBLAS_STATUS_ARCH_MISMATCH."; break;}

            case HIPBLAS_STATUS_MAPPING_ERROR:{std::cerr << "HIPBLAS_STATUS_MAPPING_ERROR."; break;}

            case HIPBLAS_STATUS_EXECUTION_FAILED:{std::cerr << "HIPBLAS_STATUS_EXECUTION_FAILED."; break;}

            case HIPBLAS_STATUS_INTERNAL_ERROR:{std::cerr << "HIPBLAS_STATUS_INTERNAL_ERROR."; break;}

            case HIPBLAS_STATUS_NOT_SUPPORTED:{std::cerr << "HIPBLAS_STATUS_NOT_SUPPORTED."; break;}

            case HIPBLAS_STATUS_UNKNOWN:{std::cerr << "HIPBLAS_STATUS_UNKNOWN."; break;}

            default:{std::cerr << "<unknown>."; break;}

        }

        std::cerr << "\n  File: "<< file << "\n  Line: "<< line <<std::endl;

        exit(EXIT_FAILURE);

    }


}




void gpu_assert(const hipError_t &code, const char *file, const size_t &line){

   if (code != hipSuccess){
       std::cerr << "ste::gpu_assert failed.\n  Error: " << hipGetErrorString(code) << "\n  File: " << file << "\n  Line: " << line << std::endl;
   }

}

} //namespace ste



//void gpuErrchk(hipError_t code){gpuAssert(code , __FILE__ , __LINE__);}


