#include "CUDA_global.h"

namespace ste {

void CUBLAs_assert(const hipblasStatus_t &code, const char *file, const size_t &line){

    if(code != HIPBLAS_STATUS_SUCCESS){
        std::cerr <<  "CUBLAS error.\nError code: ";

        switch(code){
            case HIPBLAS_STATUS_SUCCESS:{std::cerr << "HIPBLAS_STATUS_SUCCESS."; break;}

            case HIPBLAS_STATUS_NOT_INITIALIZED:{std::cerr << "HIPBLAS_STATUS_NOT_INITIALIZED."; break;}

            case HIPBLAS_STATUS_ALLOC_FAILED:{std::cerr << "HIPBLAS_STATUS_ALLOC_FAILED."; break;}

            case HIPBLAS_STATUS_INVALID_VALUE:{std::cerr << "HIPBLAS_STATUS_INVALID_VALUE."; break;}

            case HIPBLAS_STATUS_ARCH_MISMATCH:{std::cerr << "HIPBLAS_STATUS_ARCH_MISMATCH."; break;}

            case HIPBLAS_STATUS_MAPPING_ERROR:{std::cerr << "HIPBLAS_STATUS_MAPPING_ERROR."; break;}

            case HIPBLAS_STATUS_EXECUTION_FAILED:{std::cerr << "HIPBLAS_STATUS_EXECUTION_FAILED."; break;}

            case HIPBLAS_STATUS_INTERNAL_ERROR:{std::cerr << "HIPBLAS_STATUS_INTERNAL_ERROR."; break;}

            case HIPBLAS_STATUS_NOT_SUPPORTED:{std::cerr << "HIPBLAS_STATUS_NOT_SUPPORTED."; break;}

            case HIPBLAS_STATUS_UNKNOWN:{std::cerr << "HIPBLAS_STATUS_UNKNOWN."; break;}

            default:{std::cerr << "<unknown>."; break;}

        }

        std::cerr << "\n  File: "<< file << "\n  Line: "<< line <<std::endl;

        exit(EXIT_FAILURE);

    }


}




void gpu_assert(const hipError_t &code, const char *file, const size_t &line){

   if (code != hipSuccess){
       std::cerr << "ste::gpu_assert failed.\n  Error: " << hipGetErrorString(code) << "\n  File: " << file << "\n  Line: " << line << std::endl;
   }

}

} //namespace ste



//void gpuErrchk(hipError_t code){gpuAssert(code , __FILE__ , __LINE__);}


