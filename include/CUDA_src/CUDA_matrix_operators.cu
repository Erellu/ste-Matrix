#include "CUDA_matrix_operators.h"

namespace ste {

std::vector<float> CUDA_mult_MAT(const std::vector<float> &data_1 , const size_t &data_1_rows, const size_t &data_1_columns,
                                 const std::vector<float> &data_2 , const size_t &data_2_rows, const size_t &data_2_columns){

    (void)data_2_rows; //This is passed in argument only for clarity.

    hipblasHandle_t handle;

    ste_cublas_error_check(hipblasCreate(&handle));

    std::vector<float> result(data_1_rows * data_2_columns);

    /*----------------------------------------------------------------------------------------------*/

    float* GPU_data_1 = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_data_1) , data_1.size()*sizeof(float)));
    ste_gpu_error_check(hipMemcpy(GPU_data_1, data_1.data(), data_1.size()*sizeof(float), hipMemcpyHostToDevice));

    float* GPU_data_2 = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_data_2),data_2.size()*sizeof(float)));
    ste_gpu_error_check(hipMemcpy(GPU_data_2, data_2.data(), data_2.size()*sizeof(float), hipMemcpyHostToDevice));

    float* GPU_result = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_result) , result.size()*sizeof(float)));

    /*----------------------------------------------------------------------------------------------*/

    //hipblasSgemm(handle , operation , operation , m , n , k , alpha , A , lda , B , ldb , beta , C , ldc

    //(m X n) * (n X k) -> (m X k)

    //C = (alpha*A) * B + (beta*C)

    constexpr float alpha = 1.f; //Needs to be defined as a variable as it can be either a host or a device pointer (type float* in argument)
    constexpr float beta = 0.f;

    ste_cublas_error_check(
                hipblasSgemm(handle , HIPBLAS_OP_N , HIPBLAS_OP_N,
                           data_2_columns , data_1_rows ,data_1_columns,
                           &alpha , GPU_data_2 , data_2_columns,
                           GPU_data_1 , data_1_columns,
                           &beta , GPU_result , data_2_columns)
                );


    ste_gpu_error_check(hipMemcpy(result.data() , GPU_result , result.size() * sizeof(float) , hipMemcpyDeviceToHost));

    ste_gpu_error_check(hipFree(GPU_data_1));

    ste_gpu_error_check(hipFree(GPU_data_2));

    ste_gpu_error_check(hipFree(GPU_result));

    ste_cublas_error_check(hipblasDestroy(handle));

    return result;

}

std::vector<double> CUDA_mult_MAT(const std::vector<double> &data_1 , const size_t &data_1_rows, const size_t &data_1_columns,
                                  const std::vector<double> &data_2 , const size_t &data_2_rows, const size_t &data_2_columns){

    (void)data_2_rows; //This is passed in argument only for clarity.

    hipblasHandle_t handle;

    ste_cublas_error_check(hipblasCreate(&handle));

    std::vector<double> result(data_1_rows * data_2_columns);

    /*----------------------------------------------------------------------------------------------*/

    double* GPU_data_1 = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_data_1) , data_1.size()*sizeof(double)));
    ste_gpu_error_check(hipMemcpy(GPU_data_1, data_1.data(), data_1.size()*sizeof(float), hipMemcpyHostToDevice));

    double* GPU_data_2 = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_data_2),data_2.size()*sizeof(double)));
    ste_gpu_error_check(hipMemcpy(GPU_data_2, data_2.data(), data_2.size()*sizeof(float), hipMemcpyHostToDevice));

    double* GPU_result = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_result) , result.size()*sizeof(double)));

    /*----------------------------------------------------------------------------------------------*/

    //hipblasSgemm(handle , operation , operation , m , n , k , alpha , A , lda , B , ldb , beta , C , ldc

    //(m X n) * (n X k) -> (m X k)

    //C = (alpha*A) * B + (beta*C)

    constexpr double alpha = 1.f; //Needs to be defined as a variable as it can be either a host or a device pointer (type float* in argument)
    constexpr double beta = 0.f;

    ste_cublas_error_check(
                hipblasDgemm(handle , HIPBLAS_OP_N , HIPBLAS_OP_N,
                           data_2_columns , data_1_rows ,data_1_columns,
                           &alpha , GPU_data_2 , data_2_columns,
                           GPU_data_1 , data_1_columns,
                           &beta , GPU_result , data_2_columns)
                );


    ste_gpu_error_check(hipMemcpy(result.data() , GPU_result , result.size() * sizeof(float) , hipMemcpyDeviceToHost));

    ste_gpu_error_check(hipFree(GPU_data_1));

    ste_gpu_error_check(hipFree(GPU_data_2));

    ste_gpu_error_check(hipFree(GPU_result));

    ste_cublas_error_check(hipblasDestroy(handle));

    return result;

}


std::vector<float> CUDA_transpose(const std::vector<float> &data , const size_t &rows , const size_t &columns){

    hipblasHandle_t handle;
    ste_cublas_error_check(hipblasCreate(&handle));

    std::vector<float> result(data.size());

    float* GPU_data = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_data) , data.size()*sizeof(float)));

    float* GPU_data_clone = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_data_clone) , data.size()*sizeof(float)));
    ste_gpu_error_check(hipMemcpy(GPU_data_clone, data.data(), data.size()*sizeof(float), hipMemcpyHostToDevice));

    constexpr float alpha = 1.0;
    constexpr float beta = 0.0;

    hipblasSgeam( handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, columns, &alpha, GPU_data_clone, columns , &beta, GPU_data_clone, rows, GPU_data, rows );


    ste_gpu_error_check(hipMemcpy(result.data() , GPU_data , result.size() * sizeof(float) , hipMemcpyDeviceToHost));

    ste_gpu_error_check(hipFree(GPU_data));
    ste_gpu_error_check(hipFree(GPU_data_clone));

    ste_cublas_error_check(hipblasDestroy(handle));


    return result;
}

std::vector<double> CUDA_transpose(const std::vector<double> &data , const size_t &rows , const size_t &columns){

    hipblasHandle_t handle;
    ste_cublas_error_check(hipblasCreate(&handle));

    std::vector<double> result(data.size());

    double* GPU_data = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_data) , data.size()*sizeof(double)));

    double* GPU_data_clone = NULL;
    ste_gpu_error_check(hipMalloc(reinterpret_cast<void**>(&GPU_data_clone) , data.size()*sizeof(double)));
    ste_gpu_error_check(hipMemcpy(GPU_data_clone, data.data(), data.size()*sizeof(double), hipMemcpyHostToDevice));

    constexpr double alpha = 1.0;
    constexpr double beta = 0.0;

    hipblasDgeam( handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, columns, &alpha, GPU_data_clone, columns , &beta, GPU_data_clone, rows, GPU_data, rows );

    ste_gpu_error_check(hipMemcpy(result.data() , GPU_data , result.size() * sizeof(double) , hipMemcpyDeviceToHost));

    ste_gpu_error_check(hipFree(GPU_data));
    ste_gpu_error_check(hipFree(GPU_data_clone));

    ste_cublas_error_check(hipblasDestroy(handle));

    return result;

}


}//namespace ste




